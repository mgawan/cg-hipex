#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include "./include/hipex.hpp"

using namespace cooperative_groups;
const int nTPB = 32;
__global__ void my_reduce_kernel(int *data){

  //   __shared__ int sdata[nTPB];
    // task 1a: create a proper thread block group below
    auto g1 = this_thread_block();
    auto g2 = tiled_partition(g1,8);
    // auto g2 = tiled_partition<4>(g1);
  //   int test_b = 0;
  //   if(g1.thread_rank() == 31 || g1.thread_rank() == 30) {test_b = 12;}
  
  //   auto ballot = hipex::ballot(g1,test_b);
    // auto ballot = g2.ballot(test_b);
  
    // printf("thread id(g1): %d, thread id(g2): %d, meta_group_size(g1):%d, meta_grp_size(g2):%d, \
    // meta_group_id(g1):%d, meta_group_id(g2):%d my thread id:%d blocksize:%d, hipex:metasize(g2):%d\n",g1.thread_rank(), g2.thread_rank(), g1.size(), \
    // g2.size(), g1.thread_rank(), g2.thread_rank(), threadIdx.x, blockDim.x, hipex::meta_group_size(g2));
  
    printf("threadId:%d, meta_group_size:%d, group_rank:%d\n",threadIdx.x, hipex::meta_group_size(g2), hipex::meta_group_rank(g2));
    // printf("threadId:%d, meta_group_size:%d, group_rank:%d\n",threadIdx.x, g2.meta_group_size(), g2.meta_group_rank());
  }
  

int main(){

  int *data;
  hipMallocManaged(&data, nTPB*sizeof(data[0]));
  for (int i = 0; i < nTPB; i++) data[i] = 1;
  my_reduce_kernel<<<1,nTPB>>>(data);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) printf("cuda error: %s\n", hipGetErrorString(err));
}