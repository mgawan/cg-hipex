#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <stdint.h>
#include <iostream>

#define GRPSIZE 32
using namespace cooperative_groups;
const int nTPB = 32;
// __device__ int reduce(thread_group g, int *x, int val) { 
//   int lane = g.thread_rank();
//   for (int i = g.size()/2; i > 0; i /= 2) {
//     x[lane] = val;       g.sync();
//     if (lane < i) val += x[lane + i];  g.sync();
//   }
//   if (g.thread_rank() == 0) printf("group partial sum: %d\n", val);
//   return val;
// }
namespace hipex{

    __device__ uint32_t reduce(cooperative_groups::thread_group &g, uint32_t* target){
        
        int lane = (int)g.thread_rank();
        uint32_t val = target[lane];
        for(int i = g.size()/2; i > 0; i /=2){
            target[lane] = val;
            g.sync();
            if(lane < i)
                val |= target[lane+i]; 
            g.sync();
        }
        if(lane == 0)
            target[lane] = val;
        g.sync();
    }

    __device__ uint32_t ohc_id(uint32_t idx){
        uint32_t ohc_ = 0;
        ohc_ |= 1U << idx;

        return ohc_;
    }

    // template<Typename T> make it templated later, may be only uptill 32 buts for now
    __device__ uint32_t ballot(cooperative_groups::thread_group &g, int predicate){
        uint32_t ballot_ = 0;
        __shared__ uint32_t pred_sh[GRPSIZE]; // thread group sizes are limited to 32 for now
        
        assert(g.size() <= GRPSIZE);

        if(g.thread_rank() < GRPSIZE){
            pred_sh[g.thread_rank()] = 0;
            if(predicate != 0)
                pred_sh[g.thread_rank()] = ohc_id(g.thread_rank());
        }
        g.sync();
        reduce(g, pred_sh);
        ballot_ = pred_sh[0];
        return ballot_;
    }
}
__global__ void my_reduce_kernel(int *data){

//   __shared__ int sdata[nTPB];
  // task 1a: create a proper thread block group below
  auto g1 = this_thread_block();
  // auto g2 = tiled_partition<32>(g1);

  int test_b = 0;
  if(g1.thread_rank() == 31 || g1.thread_rank() == 30) {test_b = 12;}

  auto ballot = hipex::ballot(g1,test_b);
  // auto ballot = g2.ballot(test_b);

  printf("ballot from g1: %x from thread:%d\n",ballot, g1.thread_rank());

}

int main(){

  int *data;
  hipMallocManaged(&data, nTPB*sizeof(data[0]));
  for (int i = 0; i < nTPB; i++) data[i] = 1;
  my_reduce_kernel<<<1,nTPB>>>(data);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) printf("cuda error: %s\n", hipGetErrorString(err));
}